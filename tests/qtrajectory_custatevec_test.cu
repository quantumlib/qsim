// Copyright 2019 Google LLC. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "qtrajectory_testfixture.h"

#include <hipblas.h>
#include <custatevec.h>

#include "gtest/gtest.h"

#include "../lib/fuser_mqubit.h"
#include "../lib/gates_cirq.h"
#include "../lib/io.h"
#include "../lib/run_qsim.h"
#include "../lib/simulator_custatevec.h"

namespace qsim {

template <typename FP>
struct Factory {
  using fp_type = FP;
  using Simulator = qsim::SimulatorCuStateVec<fp_type>;
  using StateSpace = typename Simulator::StateSpace;

  Factory() {
    ErrorCheck(hipblasCreate(&cublas_handle));
    ErrorCheck(custatevecCreate(&custatevec_handle));
  }

  ~Factory() {
    ErrorCheck(hipblasDestroy(cublas_handle));
    ErrorCheck(custatevecDestroy(custatevec_handle));
  }

  StateSpace CreateStateSpace() const {
    return StateSpace(cublas_handle, custatevec_handle);
  }

  Simulator CreateSimulator() const {
    return Simulator(cublas_handle, custatevec_handle);
  }

  hipblasHandle_t cublas_handle;
  custatevecHandle_t custatevec_handle;
};

TEST(QTrajectoryCuStateVecTest, BitFlip) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestBitFlip<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, GenDump) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestGenDump<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, ReusingResults) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestReusingResults<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, CollectKopStat) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestCollectKopStat<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, CleanCircuit) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestCleanCircuit<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, InitialState) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestInitialState<Runner>(Factory<float>());
}

TEST(QTrajectoryCuStateVecTest, UncomputeFinalState) {
  using Fuser = MultiQubitGateFuser<IO, const Cirq::GateCirq<float>*>;
  using Runner = QSimRunner<IO, Fuser, Factory<float>>;
  TestUncomputeFinalState<Runner>(Factory<float>());
}

}  // namespace qsim

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
